
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include <cuda_runtime.h>
#include <cuda_fp16.h>
#include <cuda_bf16.h>

#include "ops/TensorOps.cuh"
#include "core/Tensor.h"

namespace OwnTensor
{

/*
void cuda_add_tensor_inplace(Tensor& lhs, const Tensor& rhs)
{
    // For simple operations, you can reuse the out-of-place kernel
    // by passing lhs as both input and output
    cuda_add_tensor(lhs, rhs, lhs);  // If your kernel supports this
}
*/


template<typename T>
__global__ void sub_kernel(const T* a, const T* b, T* output, size_t n)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        output[idx] = a[idx] - b[idx];
    }
}

template <>
__global__ void sub_kernel<__half>(const __half* a, const __half* b, __half* output, size_t n)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        output[idx] = __hsub(a[idx], b[idx]);
    }
}

template <>
__global__ void sub_kernel<__nv_bfloat16>(const __nv_bfloat16* a, const __nv_bfloat16* b, __nv_bfloat16* output, size_t n)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        output[idx] = __hsub(a[idx],b[idx]);
    }
}

void cuda_sub_tensor(const Tensor& A, const Tensor& B, Tensor& output)
{
    size_t total_elems = A.numel();
    size_t block_size = 256;
    size_t grid_size = (total_elems + block_size - 1) / block_size;
    std::cout << "Subtraction CUDA\n";

    dispatch_by_dtype(A.dtype(), [&](auto dummy)
    {
        using T = decltype(dummy);
        const T* a_ptr = A.data<T>();
        const T* b_ptr = B.data<T>();
        T* output_ptr = output.data<T>();

        sub_kernel<<<grid_size, block_size>>>(a_ptr, b_ptr, output_ptr, total_elems);

        cudaError_t err = cudaGetLastError();
        if (err != cudaSuccess) {
            throw std::runtime_error("Subtraction CUDA kernel failed: " + std::string(cudaGetErrorString(err)));
        }

        err = cudaDeviceSynchronize();
        if (err != cudaSuccess) {
            throw std::runtime_error("Subtraction CUDA kernel execution failed: " + std::string(cudaGetErrorString(err)));
        }
    });
}

/*########################################################
            TENSOR INPLACE CUDA KERNELS
##########################################################*/


template <typename T>
__global__ void sub_inplace_kernel(T* lhs, const T* rhs, size_t n)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        lhs[idx] -= rhs[idx];
    }
}

template <>
__global__ void sub_inplace_kernel<__half>(__half* lhs, const __half* rhs, size_t n)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        lhs[idx] = __hsub(lhs[idx], rhs[idx]);
    }
}

template <>
__global__ void sub_inplace_kernel<__nv_bfloat16>(__nv_bfloat16* lhs, const __nv_bfloat16* rhs, size_t n)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        lhs[idx] = __hsub(lhs[idx],rhs[idx]);
    }
}

void cuda_sub_tensor_inplace(Tensor& A, const Tensor& B)
    {
        size_t total_elems = A.numel();
        size_t block_size = 256;
        size_t grid_size = (total_elems + block_size - 1) / block_size;

        std::cout << "Addition Inplace CUDA\n";

        dispatch_by_dtype(A.dtype(), [&](auto dummy)
        {
            using T = decltype(dummy);
            T* a_ptr = A.data<T>();
            const T* b_ptr = B.data<T>();
                
            sub_inplace_kernel<<<grid_size, block_size>>>(a_ptr, b_ptr, total_elems);
            
            cudaError_t err = cudaGetLastError();
            if (err != cudaSuccess) {
                throw std::runtime_error("Addition CUDA kernel failed: " + std::string(cudaGetErrorString(err)));
            }
            
            err = cudaDeviceSynchronize();
            if (err != cudaSuccess) {
                throw std::runtime_error("Addition CUDA kernel execution failed: " + std::string(cudaGetErrorString(err)));
            }
        });
    }

}

#endif
