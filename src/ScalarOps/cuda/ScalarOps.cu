
#include <hip/hip_runtime.h>
#if defined(WITH_CUDA)
#include <cuda_runtime.h>
#include <device_launch_parameters.h>
#include <cuda_fp16.h>
#include <cuda_bf16.h>
#include <stdexcept>
#include <type_traits>
#include "core/Tensor.h"
#include "core/TensorDispatch.h"
#include "dtype/Types.h"

namespace OwnTensor {
namespace { // file-local CUDA helpers & kernels

inline int half_fmt(Dtype dt) { // 0 = numeric; 1 = fp16; 2 = bf16
    return (dt == Dtype::Float16) ? 1 : (dt == Dtype::Bfloat16 ? 2 : 0);
}

__device__ inline float dev_bf16_to_float(uint16_t b) {
    uint32_t u = ((uint32_t)b) << 16;
    return __uint_as_float(u);
}
__device__ inline uint16_t dev_float_to_bf16(float f) {
    uint32_t u = __float_as_uint(f);
    uint32_t lsb = (u >> 16) & 1u;
    u += 0x7FFFu + lsb; // RNE
    return (uint16_t)(u >> 16);
}
__device__ inline float dev_fp16_to_float(uint16_t bits) {
    __half h = *reinterpret_cast<const __half*>(&bits);
    return __half2float(h);
}
__device__ inline uint16_t dev_float_to_fp16(float f) {
    __half h = __float2half_rn(f);
    return *reinterpret_cast<uint16_t*>(&h);
}

template <typename T>
__device__ inline float ldf(const T* p, size_t i, int) { return static_cast<float>(p[i]); }

template <>
[[maybe_unused]] __device__ inline float ldf<uint16_t>(const uint16_t* p, size_t i, int fmt) {
    return (fmt == 1) ? dev_fp16_to_float(p[i])
         : (fmt == 2) ? dev_bf16_to_float(p[i])
                      : static_cast<float>(p[i]);
}

template <typename T>
__device__ inline void stf(T* p, size_t i, float v, int) { p[i] = static_cast<T>(v); }

template <>
[[maybe_unused]] __device__ inline void stf<uint16_t>(uint16_t* p, size_t i, float v, int fmt) {
    p[i] = (fmt == 1) ? dev_float_to_fp16(v)
         : (fmt == 2) ? dev_float_to_bf16(v)
                      : static_cast<uint16_t>(v);
}

inline dim3 pick_grid(size_t n, dim3 b) {
    size_t blocks = (n + b.x - 1) / b.x;
    if (blocks > 2147483647ULL) blocks = 2147483647ULL;
    return dim3(static_cast<unsigned int>(blocks));
}

inline void ckerr(const char* where) {
    cudaError_t e = cudaGetLastError();
    if (e != cudaSuccess) throw std::runtime_error(std::string(where) + ": " + cudaGetErrorString(e));
}

// ---- kernels (in-place) ----
template<typename T>
__global__ void k_add_inplace(T* d, float s, size_t n, int fmt) {
    for (size_t i = blockIdx.x*blockDim.x + threadIdx.x; i < n; i += blockDim.x*gridDim.x)
        stf<T>(d, i, ldf<T>(d, i, fmt) + s, fmt);
}
template<typename T>
__global__ void k_sub_inplace(T* d, float s, size_t n, int fmt) {
    for (size_t i = blockIdx.x*blockDim.x + threadIdx.x; i < n; i += blockDim.x*gridDim.x)
        stf<T>(d, i, ldf<T>(d, i, fmt) - s, fmt);
}
template<typename T>
__global__ void k_mul_inplace(T* d, float s, size_t n, int fmt) {
    for (size_t i = blockIdx.x*blockDim.x + threadIdx.x; i < n; i += blockDim.x*gridDim.x)
        stf<T>(d, i, ldf<T>(d, i, fmt) * s, fmt);
}
template<typename T>
__global__ void k_div_inplace(T* d, float s, size_t n, int fmt) {
    for (size_t i = blockIdx.x*blockDim.x + threadIdx.x; i < n; i += blockDim.x*gridDim.x)
        stf<T>(d, i, ldf<T>(d, i, fmt) / s, fmt);
}

// ---- kernels (copy) ----
template<typename T>
__global__ void k_add_copy(const T* a, T* o, float s, size_t n, int fmt) {
    for (size_t i = blockIdx.x*blockDim.x + threadIdx.x; i < n; i += blockDim.x*gridDim.x)
        stf<T>(o, i, ldf<T>(a, i, fmt) + s, fmt);
}
template<typename T>
__global__ void k_sub_copy(const T* a, T* o, float s, size_t n, int fmt) {
    for (size_t i = blockIdx.x*blockDim.x + threadIdx.x; i < n; i += blockDim.x*gridDim.x)
        stf<T>(o, i, ldf<T>(a, i, fmt) - s, fmt);
}
template<typename T>
__global__ void k_mul_copy(const T* a, T* o, float s, size_t n, int fmt) {
    for (size_t i = blockIdx.x*blockDim.x + threadIdx.x; i < n; i += blockDim.x*gridDim.x)
        stf<T>(o, i, ldf<T>(a, i, fmt) * s, fmt);
}
template<typename T>
__global__ void k_div_copy(const T* a, T* o, float s, size_t n, int fmt) {
    for (size_t i = blockIdx.x*blockDim.x + threadIdx.x; i < n; i += blockDim.x*gridDim.x)
        stf<T>(o, i, ldf<T>(a, i, fmt) / s, fmt);
}

template<typename T>
__global__ void k_sub_copy_scalar_tensor(const T* a, T* o, float s, size_t n, int fmt) {
    for (size_t i = blockIdx.x*blockDim.x + threadIdx.x; i < n; i += blockDim.x*gridDim.x)
        stf<T>(o, i, s - ldf<T>(a, i, fmt), fmt);
}

template<typename T>
__global__ void k_div_copy_scalar_tensor(const T* a, T* o, float s, size_t n, int fmt, int* flag) {
    for (size_t i = blockIdx.x*blockDim.x + threadIdx.x; i < n; i += blockDim.x*gridDim.x) {
        if constexpr (std::is_integral_v<T>) {
            if (fmt == 0 && a[i] == (T)0) { if (flag) atomicExch(flag, 1); }
        }
        stf<T>(o, i, s / ldf<T>(a, i, fmt), fmt);
    }
}

// ---- launch helpers ----
template <typename T, typename Kernel>
inline void launch_copy(const Tensor& a, Tensor& out, double s, Kernel k) {
    const size_t n = a.numel();
    const dim3 block = dim3(256), grid = pick_grid(n, block);
    const int fmt = half_fmt(a.dtype());
    k<<<grid, block>>>(a.data<T>(), out.data<T>(), (float)s, n, fmt);
    ckerr("scalar copy");
}

template <typename T, typename Kernel>
inline void launch_inplace(Tensor& t, double s, Kernel k) {
    const size_t n = t.numel();
    const dim3 block = dim3(256), grid = pick_grid(n, block);
    const int fmt = half_fmt(t.dtype());
    k<<<grid, block>>>(t.data<T>(), (float)s, n, fmt);
    ckerr("scalar inplace");
}

} // anon namespace

// --------- public CUDA backend ---------
void cuda_add_inplace(Tensor& t, double s) {
    dispatch_by_dtype(t.dtype(), [&](auto d){ using T = decltype(d); launch_inplace<T>(t, s, k_add_inplace<T>); });
}
void cuda_sub_inplace(Tensor& t, double s) {
    dispatch_by_dtype(t.dtype(), [&](auto d){ using T = decltype(d); launch_inplace<T>(t, s, k_sub_inplace<T>); });
}
void cuda_mul_inplace(Tensor& t, double s) {
    dispatch_by_dtype(t.dtype(), [&](auto d){ using T = decltype(d); launch_inplace<T>(t, s, k_mul_inplace<T>); });
}
void cuda_div_inplace(Tensor& t, double s) {
    dispatch_by_dtype(t.dtype(), [&](auto d){ using T = decltype(d); launch_inplace<T>(t, s, k_div_inplace<T>); });
}

Tensor cuda_add_copy(const Tensor& a, double s) {
    Tensor out(a.shape(), a.dtype(), a.device(), a.requires_grad());
    dispatch_by_dtype(a.dtype(), [&](auto d){ using T = decltype(d); launch_copy<T>(a, out, s, k_add_copy<T>); });
    return out;
}
Tensor cuda_sub_copy(const Tensor& a, double s) {
    Tensor out(a.shape(), a.dtype(), a.device(), a.requires_grad());
    dispatch_by_dtype(a.dtype(), [&](auto d){ using T = decltype(d); launch_copy<T>(a, out, s, k_sub_copy<T>); });
    return out;
}
Tensor cuda_mul_copy(const Tensor& a, double s) {
    Tensor out(a.shape(), a.dtype(), a.device(), a.requires_grad());
    dispatch_by_dtype(a.dtype(), [&](auto d){ using T = decltype(d); launch_copy<T>(a, out, s, k_mul_copy<T>); });
    return out;
}
Tensor cuda_div_copy(const Tensor& a, double s) {
    Tensor out(a.shape(), a.dtype(), a.device(), a.requires_grad());
    dispatch_by_dtype(a.dtype(), [&](auto d){ using T = decltype(d); launch_copy<T>(a, out, s, k_div_copy<T>); });
    return out;
}

Tensor cuda_sub_copy_scalar_tensor(double s, const Tensor& a) {
    Tensor out(a.shape(), a.dtype(), a.device(), a.requires_grad());
    dispatch_by_dtype(a.dtype(), [&](auto d){ using T = decltype(d); launch_copy<T>(a, out, s, k_sub_copy_scalar_tensor<T>); });
    return out;
}

Tensor cuda_div_copy_scalar_tensor(double s, const Tensor& a) {
    Tensor out(a.shape(), a.dtype(), a.device(), a.requires_grad());
    dispatch_by_dtype(a.dtype(), [&](auto d){
        using T = decltype(d);
        const size_t n = a.numel();
        const dim3 block = dim3(256), grid = pick_grid(n, block);
        const int fmt = half_fmt(a.dtype());

        int host_flag = 0;
        int* dev_flag = nullptr;
        cudaMalloc(&dev_flag, sizeof(int));
        cudaMemset(dev_flag, 0, sizeof(int));

        k_div_copy_scalar_tensor<T><<<grid, block>>>(a.data<T>(), out.data<T>(), (float)s, n, fmt, dev_flag);
        ckerr("k_div_copy_scalar_tensor");

        cudaMemcpy(&host_flag, dev_flag, sizeof(int), cudaMemcpyDeviceToHost);
        cudaFree(dev_flag);

        if (host_flag) throw std::runtime_error("Division by zero in scalar / integer tensor");
    });
    return out;
}

} // namespace OwnTensor
#endif // WITH_CUDA
